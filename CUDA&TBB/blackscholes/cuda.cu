#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include "main.h"

const double RISK_FREE = 0.02;
const double VOLATILITY = 0.30;

__device__ double blackScholesFormulaCUDA(double S, double K, double T)
{
    // Reemplazar funciones de C++ con las versiones de CUDA
    double d1 = (logf(S / K) + (RISK_FREE + 0.5 * VOLATILITY * VOLATILITY) * T) / (VOLATILITY * sqrtf(T));
    double d2 = d1 - VOLATILITY * sqrtf(T);
    double callPrice = S * 0.5 * (1 + erff(d1 / sqrtf(2))) - K * expf(-RISK_FREE * T) * 0.5 * (1 + erff(d2 / sqrtf(2)));
    return callPrice;
}

__global__ void calculatePricesCUDA(double* prices, double* results, int num_prices)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < num_prices)
    {
        results[idx] = blackScholesFormulaCUDA(prices[idx], 100, 1); // K=100, T=1
    }
}

void holaMundoCUDA(std::vector<double>& prices, std::vector<double>& results, int num_prices)
{
    double* d_prices;
    double* d_results;

    hipMalloc(&d_prices, num_prices * sizeof(double));
    hipMalloc(&d_results, num_prices * sizeof(double));

    hipMemcpy(d_prices, prices.data(), num_prices * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (num_prices + blockSize - 1) / blockSize;

    calculatePricesCUDA<<<numBlocks, blockSize>>>(d_prices, d_results, num_prices);

    hipMemcpy(results.data(), d_results, num_prices * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_prices);
    hipFree(d_results);
}
